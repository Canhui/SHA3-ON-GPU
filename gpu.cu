
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdbool.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <inttypes.h>
#include <errno.h>


#define ROTL64(x, y) (((x) << (y)) | ((x) >> (64 - (y))))

#define MSG_LEN 10
#define HASH_LEN 30
#define THREADS_PER_BLOCK 512

// Round constants
__constant__ static const uint64_t RC[24] = {
    0x0000000000000001, 0x0000000000008082, 0x800000000000808a,
    0x8000000080008000, 0x000000000000808b, 0x0000000080000001,
    0x8000000080008081, 0x8000000000008009, 0x000000000000008a,
    0x0000000000000088, 0x0000000080008009, 0x000000008000000a,
    0x000000008000808b, 0x800000000000008b, 0x8000000000008089,
    0x8000000000008003, 0x8000000000008002, 0x8000000000000080, 
    0x000000000000800a, 0x800000008000000a, 0x8000000080008081,
    0x8000000000008080, 0x0000000080000001, 0x8000000080008008
};


// Rotation offsets
__constant__ static const int r[24] = {
    1,  3,  6,  10, 15, 21, 28, 36, 45, 55, 2,  14, 
    27, 41, 56, 8,  25, 43, 62, 18, 39, 61, 20, 44
};



__constant__ static const int piln[24] = {
    10, 7,  11, 17, 18, 3, 5,  16, 8,  21, 24, 4, 
    15, 23, 19, 13, 12, 2, 20, 14, 22, 9,  6,  1 
};



// Updates the state with 24 rounds
__device__ void keccakf(uint64_t *state){
    int i, j;
    uint64_t temp, C[5];

    for (int round = 0; round < 24; round++) {
        // Theta
        for (i = 0; i < 5; i++) {
            C[i] = state[i] ^ state[i + 5] ^ state[i + 10] ^ state[i + 15] ^ state[i + 20];
        }

        for (i = 0; i < 5; i++) {
            temp = C[(i + 4) % 5] ^ ROTL64(C[(i + 1) % 5], 1);
            for (j = 0; j < 25; j += 5) {
                state[j + i] ^= temp;
            }
        }

        // Rho Pi
        temp = state[1];
        for (i = 0; i < 24; i++) {
            j = piln[i];
            C[0] = state[j];
            state[j] = ROTL64(temp, r[i]);
            temp = C[0];
        }

        //  Chi
        for (j = 0; j < 25; j += 5) {
            for (i = 0; i < 5; i++) {
                C[i] = state[j + i];
            }
            for (i = 0; i < 5; i++) {
                state[j + i] ^= (~C[(i + 1) % 5]) & C[(i + 2) % 5];
            }
        }

        //  Iota
        state[0] ^= RC[round];
    }
}



__global__ void keccak__offset(uint8_t *message_, unsigned long numbytes){
	int message_len = MSG_LEN; 
	uint64_t state[25];    
    uint8_t temp[144];
    int rsize = 136;            // 200 - 2 * 32
    int rsize_byte = 17;        // rsize / 8
    
    uint8_t message[MSG_LEN];

    unsigned long offset = (blockIdx.x * THREADS_PER_BLOCK * MSG_LEN) + (threadIdx.x * MSG_LEN);
  	if (offset >= numbytes) {  return; }

  	memcpy(message, &message_[offset], MSG_LEN);

  	memset(state, 0, sizeof(state));

  	// for ( ; message_len >= rsize; message_len -= rsize, (uint8_t *)message += rsize) {
        for (int i = 0; i < rsize_byte; i++) {
            state[i] ^= ((uint64_t *) message)[i];
        }
        keccakf(state);
    // }

    // Calculating the last state block and padding the result
    memcpy(temp, message, message_len);
    temp[message_len++] = 1;
    memset(temp + message_len, 0, rsize - message_len);
    temp[rsize - 1] |= 0x80;

    for (int i = 0; i < rsize_byte; i++) {
        state[i] ^= ((uint64_t *) temp)[i];
    }

    keccakf(state);
    __syncthreads();
}


void hashdemo(uint8_t *message_, unsigned long numbytes){
  uint8_t *message;
  // uint8_t *ctx_key_d, *ctx_enckey_d;

// RC[24]
  hipMemcpyToSymbol(HIP_SYMBOL(RC), RC, sizeof(uint64_t)*24);
  hipMemcpyToSymbol(HIP_SYMBOL(r), r, sizeof(int)*24);
  hipMemcpyToSymbol(HIP_SYMBOL(piln), piln, sizeof(int)*24);



  hipMalloc((void**)&message, numbytes);


  hipMemcpy(message, message_, numbytes, hipMemcpyHostToDevice);

  dim3 dimBlock(ceil((double)numbytes / (double)(THREADS_PER_BLOCK * MSG_LEN)));
  dim3 dimGrid(THREADS_PER_BLOCK);


  keccak__offset<<<dimBlock, dimGrid>>>(message, numbytes);


  hipFree(message);
  // cudaFree(ctx_key_d);
  // cudaFree(ctx_enckey_d);
}

__global__ void GPU_init() { }


int main(){

  // open file
  FILE *file;
  uint8_t *buf; // file buffer
  unsigned long numbytes;
  char *fname;
  clock_t start, enc_time, dec_time, end;
  int mili_sec, i;
  int padding;
  //key: 00 01 02 03 04 05 06 07 08 09 0a 0b 0c 0d 0e 0f 10 11 12 13 14 15 16 17 18 19 1a 1b 1c 1d 1e 1f
    /* create a key vector */
  uint8_t key[32];

  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (error_id != hipSuccess){
    printf("Error: %s\n", hipGetErrorString(error_id));
    printf("Exiting...\n");
    exit(EXIT_FAILURE);
  }

  // This function call returns 0 if there are no CUDA capable devices.
  if (deviceCount == 0){
    printf("There are no available device(s) that support CUDA\n");
    exit(EXIT_FAILURE);
  }


  // handle txt file
  fname = "input.txt";  
  file = fopen(fname, "r");
  if (file == NULL) {printf("File %s doesn't exist\n", fname); exit(1); }
  printf("Opened file %s\n", fname);
  fseek(file, 0L, SEEK_END);
  numbytes = ftell(file);
  printf("Size is %lu\n", numbytes);

  // copy file into memory
  fseek(file, 0L, SEEK_SET);
  buf = (uint8_t*)calloc(numbytes, sizeof(uint8_t));
  if(buf == NULL) exit(1);
  if (fread(buf, 1, numbytes, file) != numbytes)
  {
    printf("Unable to read all bytes from file %s\n", fname);
    exit(EXIT_FAILURE);
  }
  fclose(file);


  // this is to force nvcc to put the gpu initialization here
  GPU_init<<<1, 1>>>();

  // encryption
  start = clock();
  hashdemo(buf, numbytes);
  end = clock();
  printf("time used:%f\n",  (double)(end - start) / CLOCKS_PER_SEC);
  printf("CPU encryption throughput: %f bytes/second\n",  (double)(numbytes) / ((double)(end - start) / CLOCKS_PER_SEC));


  free(buf);
  return EXIT_SUCCESS;
}